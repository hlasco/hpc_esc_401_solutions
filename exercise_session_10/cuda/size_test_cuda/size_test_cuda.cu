// Using CUDA device to calculate pi
#include <stdio.h>
#include <hip/hip_runtime.h>
extern "C" double getTime(void);

#define NBIN 1000000000  // Number of bins

// Kernel that executes on the CUDA device
__global__ void cal_pi(double *sum, int nbin, double step, int nthreads, int nblocks) {
	int i;
	double x;
	int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
	for (i=idx; i< nbin; i+=nthreads*nblocks) {
		x = (i+0.5)*step;
		sum[idx] += 4.0/(1.0+x*x);
	}
}

// Main routine that executes on the host
int main(void) {
	int NUM_BLOCK[8] = {60, 120, 180, 240, 300, 360, 420, 600};
	int NUM_THREAD[10] = {16, 32, 48, 64, 80, 96, 112, 128, 144, 160};
	for (int i = 0; i<8; i++){
		for (int j = 0; j<10; j++){

	dim3 dimGrid(NUM_BLOCK[i],1,1);  // Grid dimensions
	dim3 dimBlock(NUM_THREAD[j],1,1);  // Block dimensions
	double *sumHost, *sumDev;  // Pointer to host & device arrays
	double pi = 0;
	int tid;

	double step = 1.0/NBIN;  // Step size
	size_t size = NUM_BLOCK[i]*NUM_THREAD[j]*sizeof(double);  //Array memory size
	sumHost = (double *)malloc(size);  //  Allocate array on host
	hipMalloc((void **) &sumDev, size);  // Allocate array on device
   	double start = getTime();

	// Initialize array in device to 0
	hipMemset(sumDev, 0, size);
	// Do calculation on device
	cal_pi <<<dimGrid, dimBlock>>> (sumDev, NBIN, step, NUM_THREAD[j], NUM_BLOCK[i]); // call CUDA kernel
	// Retrieve result from device and store it in host array
	hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);
	for(tid=0; tid<NUM_THREAD[j]*NUM_BLOCK[i]; tid++)
		pi += sumHost[tid];
	pi *= step;

	// Print results
	double delta = getTime() - start;
	printf("BLOCKS=%d, THREADS=%d, PI = %.16g computed in %.4g seconds\n", NUM_BLOCK[i], NUM_THREAD[j], pi, delta);
	// Cleanup
	free(sumHost);
	hipFree(sumDev);
}}	
	return 0;
}

